#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>

using namespace cv;
#define DIM 20000

struct hipComplex {
	double r;
	double i;
};

__device__ void julia(int x, int y, int& juliaValue) {
	const double scale = 1.5;
	double jx = scale * (double)(DIM / 2 - x) / (DIM / 2);
	double jy = scale * (double)(DIM / 2 - y) / (DIM / 2);
	hipComplex c;
	c.r = -0.8;
	c.i = 0.156;
	hipComplex a;
	hipComplex b;
	a.r = jx;
	a.i = jy;

	int i = 0;
	for (i = 0; i<200; i++) {
		b.r = a.r;
		b.i = a.i;
		a.r = b.r * b.r - b.i * b.i + c.r;
		a.i = b.i * b.r + b.r * b.i + c.i;
		if (a.r * a.r + a.i * a.i > 5000){
			juliaValue = 1;
			break;
		}
		else
			juliaValue = 0;
	}
}

__global__ void generateJuliaImage(unsigned char *dataPtr) {
	// map from threadIdx/BlockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	// now calculate the value at that position
	int juliaValue;
	julia(x, y, juliaValue);
	dataPtr[offset] = 255 * juliaValue;
}

int main(void)
{
	Mat cuImage(DIM, DIM, CV_8UC1);

	unsigned char *dev_cuImage;
	dim3 grid(DIM, DIM);
	int err = hipMalloc((void**)&dev_cuImage, DIM*DIM*sizeof(char));
	generateJuliaImage<<<grid, 1 >>>(dev_mImage);
	err = hipMemcpy(cuImage.data, dev_cuImage, DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
	err = hipFree(dev_cuImage);

	printf("finished, press any key to exit ...\n");
	imwrite("C:\\Users\\huyix\\Desktop\\cudaJulia.jpg", cuImage);
	
    return 0;
}
